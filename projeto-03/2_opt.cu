#include "hip/hip_runtime.h"
#include <omp.h>
#include <iostream>
#include <algorithm>    // std::random_shuffle
#include <vector>       // std::vector
#include <random>

#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include "math.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

using namespace std;

__host__ __device__ double dist(float p1x,float p1y,float p2x,float p2y){
    return sqrt(pow(p1x - p2x,2) + pow(p1y - p2y,2));
    }

/*double path_dist(vector<int> seq, place *points,int n){
    double d = dist(points[seq[n-1]], points[seq[0]]);
    for (int i=0 ;i < (n-1); i++){
        d += dist(points[seq[i]], points[seq[i+1]]);
        }
    return d;
    }*/

__device__ void swap_2opt(vector<int> aux,int i, int j){

}

__global__ void pre_calc(float *x,float *y,double *dist_matrix,int n){
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    //printf("%f \n", dist(x[i],y[i],x[j],y[j]));
    

    if ((i < n) && (j < n)){
        dist_matrix[i*n +j] = dist(x[i],y[i],x[j],y[j]);
    }
}

__global__ void solver(double *dist_matrix,int *all_seq,double *dis_calc,int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int begin = (i*n)+1;
    int end = begin +n -2; //-1 begin esta começando no 1 e mais -1 para acabar no final da lista não no começo do proximo

    hiprandState st;
    hiprand_init(0, i, 0, &st);

    for(int j=begin;j< end;j++){

        int place = (int) ((end-j) * hiprand_uniform(&st) + j);

        int tmp;
        tmp = all_seq[begin];
        all_seq[begin] = all_seq[place];
        all_seq[place] = tmp;
    }

    double my_dist = 0;

    for(int j=begin;j< end;j++){
        my_dist += dist_matrix[(all_seq[j]*n+all_seq[j+1])];
    }

    dis_calc[i] = my_dist;


    
}

int main(){
    const int maximum = 1;
    int n;
    cin >> n;

    thrust::host_vector<float> host_pointx(n);
    thrust::host_vector<float> host_pointy(n);

    thrust::device_vector<float> pointx(n);
    thrust::device_vector<float> pointy(n);
    

    for (int i=0 ;i < n; i++){
        float x;
        float y;

        cin >> x;
        cin >> y;

        

        host_pointx[i] = x;
        host_pointy[i] = y;
        
    }

    pointx = host_pointx;
    pointy = host_pointy;



    thrust::device_vector<double> dist_matrix(n*n);

    dim3 blocks(ceil(1024.0/n), ceil(1024.0/n), 1);
    dim3 th(32, 32, 1);

    

    pre_calc<<<blocks,th>>>(
        thrust::raw_pointer_cast(pointx.data()),
        thrust::raw_pointer_cast(pointy.data()),
        thrust::raw_pointer_cast(dist_matrix.data()), 
        n
        );
    
    //int all_seq_size = n*maximum*1024;
    int all_seq_size = n*3;
    
    thrust::host_vector<int> all_seq_host(all_seq_size);
    thrust::device_vector<int> all_seq(all_seq_size);
    thrust::device_vector<double> dis_calc(maximum* 3);





    for(int i=0;i<all_seq_size;i+=n){
        for (int j=0;j<n;j++){
            all_seq_host[i+j] = j;            
        }
    }
    
    //cout << "AAAAAAAAA" << endl;
    all_seq = all_seq_host;

    
    solver<<<1,3>>>(
        thrust::raw_pointer_cast(dist_matrix.data()),
        thrust::raw_pointer_cast(all_seq.data()),
        thrust::raw_pointer_cast(dis_calc.data()),
        n
        );

    thrust::host_vector<double> host_dis_calc(maximum* 3);
    thrust::device_vector<double> best_seq(n);
    thrust::host_vector<double> host_best_seq(n);

    host_dis_calc = dis_calc;
    int index_mult = 0;

    double best = host_dis_calc[0];
    for (int i =1;i<= 3;i++){
        if(host_dis_calc[0] < best){
            best = host_dis_calc[0];
            index_mult = i;
        }
    }

    thrust::copy_n(all_seq.begin()+(1*index_mult),n,best_seq.begin());

    
    for (auto i = all_seq.begin(); i != all_seq.end(); i++) {
        cout << *i << " "; // este acesso é lento! -- GPU
    }
    cout << endl;
    for (auto i = dis_calc.begin(); i != dis_calc.end(); i++) {
        cout << *i << " "; // este acesso é lento! -- GPU
    }
    cout << endl;
    for (auto i = best_seq.begin(); i != best_seq.end(); i++) {
        cout << *i << " "; // este acesso é lento! -- GPU
    }

    /*
    solver(points,n);
    cout << path_dist(solution, points,n) << " 0" << endl;

    for(int i=0 ;i < n; i++){
        cout << solution[i] << " ";
    }
    cout << endl;
    */

    return 0;
}