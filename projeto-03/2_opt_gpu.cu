#include "hip/hip_runtime.h"
#include <omp.h>
#include <iostream>
#include <algorithm>    // std::random_shuffle
#include <vector>       // std::vector
#include <random>

#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include "math.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

using namespace std;

__device__ double dist(float p1x,float p1y,float p2x,float p2y){
    return sqrt(pow(p1x - p2x,2) + pow(p1y - p2y,2));
    }

/*double path_dist(vector<int> seq, place *points,int n){
    double d = dist(points[seq[n-1]], points[seq[0]]);
    for (int i=0 ;i < (n-1); i++){
        d += dist(points[seq[i]], points[seq[i+1]]);
        }
    return d;
    }*/



__device__ double find_dist(int begin,int end, double *dist_matrix, int *all_seq,int n){
    double my_dist = 0;
    for(int j=begin;j< end;j++){//calc final dist
        my_dist += dist_matrix[(all_seq[j]*n+all_seq[j+1])];
    }
    return my_dist;
}

__device__ void opt_swap(int p1,int p2, int *vector){
    int tmp;
    tmp = vector[p1];
    vector[p1] = vector[p2];
    vector[p2] = tmp;
}


__global__ void pre_calc(float *x,float *y,double *dist_matrix,int n){
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    //printf("%f \n", dist(x[i],y[i],x[j],y[j]));
    

    if ((i < n) && (j < n)){
        dist_matrix[i*n +j] = dist(x[i],y[i],x[j],y[j]);
    }
}

__global__ void solver(double *dist_matrix,int *all_seq,double *dis_calc,int n,int total_iter,int t_work = 1){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = threadIdx.x;
    extern __shared__ int s[];
    if (i < total_iter){

        int begin = (i*n);
        int last = begin +n -1; // (inclusive end)

        //init seq in all_seq
        int tmp_counter = 0;
        for(int j=begin;j <= last;j++){
            all_seq[j] = tmp_counter;
            s[idx*n+tmp_counter] = tmp_counter;
            tmp_counter++;
        }
        //printf("AAA: %i \n",all_seq[i+1]);
        



        hiprandState st;
        double t_best = find_dist(begin,last, dist_matrix, all_seq,n); // cant use numeric_limits<double>::max()    ):
        for(int t=0;t<t_work;t++){

            

            //suflle seq
            hiprand_init(0, i+t, 0, &st);
            for(int j=begin+1;j < last;j++){
                int place = (int) ((last-j-1) * hiprand_uniform(&st) + j);
                opt_swap(j,place,all_seq);
            }
            
            bool improved = true;
            double current_best = find_dist(begin,last, dist_matrix, all_seq,n);
            while (improved){
                improved = false;
                for (int i = begin+1; i < last; i++){
                    for (int j = i+1; j < last; j++){
                    opt_swap(i,j,all_seq);
                    double possibel_best = find_dist(begin,last, dist_matrix, all_seq,n);
                    if (possibel_best < current_best){
                        improved = true;
                        current_best = possibel_best;
                    }
                    else{
                        opt_swap(i,j,all_seq);//swap back
                    }

                    }
                    
                }
            
            
            }

            if (current_best < t_best) t_best=current_best;
            
            
        }

        dis_calc[i] = t_best;
    
    }


    
}

int main(){
    const int total_iter = 5;
    const int max_th = 1024;
    const int max_blocks = ceil((float) total_iter/max_th);    
    
    int n;
    cin >> n;

    thrust::host_vector<float> host_pointx(n);
    thrust::host_vector<float> host_pointy(n);

    thrust::device_vector<float> pointx(n);
    thrust::device_vector<float> pointy(n);
    

    for (int i=0 ;i < n; i++){
        float x;
        float y;

        cin >> x;
        cin >> y;

        

        host_pointx[i] = x;
        host_pointy[i] = y;
        
    }

    pointx = host_pointx;
    pointy = host_pointy;



    thrust::device_vector<double> dist_matrix(n*n);

    dim3 blocks(ceil(n/32.0), ceil(n/32.0), 1);
    dim3 th(32, 32, 1);

    

    pre_calc<<<blocks,th>>>(
        thrust::raw_pointer_cast(pointx.data()),
        thrust::raw_pointer_cast(pointy.data()),
        thrust::raw_pointer_cast(dist_matrix.data()), 
        n
        );
    

    int all_seq_size = n*total_iter;
    thrust::device_vector<int> all_seq(all_seq_size);
    thrust::device_vector<double> dis_calc(total_iter);

    solver<<<max_blocks,max_th, 1024*n*sizeof(int)>>>(
        thrust::raw_pointer_cast(dist_matrix.data()),
        thrust::raw_pointer_cast(all_seq.data()),
        thrust::raw_pointer_cast(dis_calc.data()),
        n,
        total_iter
        );

    

    thrust::device_vector<double>::iterator iter = thrust::min_element(dis_calc.begin(), dis_calc.end());
      
    unsigned int position = iter - dis_calc.begin();
    double best = *iter;
      
    thrust::host_vector<double> host_best_seq(all_seq.begin()+position*n,all_seq.begin()+position*n+n);

    cout << best <<endl;
    for (auto i = host_best_seq.begin(); i != host_best_seq.end(); i++) {
        cout << *i << " ";
    }
    
    
    //DEBUG
    /*
    cout << best <<endl;
    for (auto i = all_seq.begin(); i != all_seq.end(); i++) {
        cout << *i << " "; // este acesso é lento! -- GPU
    }
    cout << endl;
    for (auto i = dis_calc.begin(); i != dis_calc.end(); i++) {
        cout << *i << " "; // este acesso é lento! -- GPU
    }
    cout << endl;
    for (auto i = host_best_seq.begin(); i != host_best_seq.end(); i++) {
        cout << *i << " "; // este acesso é lento! -- GPU
    }
    /**/
    

    return 0;
}