#include "hip/hip_runtime.h"
#include <omp.h>
#include <iostream>
#include <algorithm>    // std::random_shuffle
#include <vector>       // std::vector
#include <random>

#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include "math.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

using namespace std;

__host__ __device__ double dist(float p1x,float p1y,float p2x,float p2y){
    return sqrt(pow(p1x - p2x,2) + pow(p1y - p2y,2));
    }

/*double path_dist(vector<int> seq, place *points,int n){
    double d = dist(points[seq[n-1]], points[seq[0]]);
    for (int i=0 ;i < (n-1); i++){
        d += dist(points[seq[i]], points[seq[i+1]]);
        }
    return d;
    }*/

__global__ void pre_calc(float *x,float *y,double *dist_matrix,int n){
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    //printf("%f \n", dist(x[i],y[i],x[j],y[j]));
    

    if ((i < n) && (j < n)){
        dist_matrix[i*n +j] = dist(x[i],y[i],x[j],y[j]);
    }
}

__global__ void solver(double *dist_matrix,int *all_seq,double *dis_calc,int n,int total_iter){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < total_iter){
        int begin = (i*n);
        int end = begin +n -1; //-1 para acabar no final da lista não no começo do proximo (end inclusivo)

        hiprandState st;
        hiprand_init(0, i, 0, &st);

        for(int j=begin+1;j < end;j++){

            int place = (int) ((end-j-1) * hiprand_uniform(&st) + j);

            int tmp;
            tmp = all_seq[j];
            all_seq[j] = all_seq[place];
            all_seq[place] = tmp;
        }

        double my_dist = 0;

        for(int j=begin;j< end;j++){
            my_dist += dist_matrix[(all_seq[j]*n+all_seq[j+1])];
        }

        dis_calc[i] = my_dist;
    }


    
}

int main(){
    const int max_blocks = 10;
    const int max_th = 1024;
    const int total_iter = 10000;
    int n;
    cin >> n;

    thrust::host_vector<float> host_pointx(n);
    thrust::host_vector<float> host_pointy(n);

    thrust::device_vector<float> pointx(n);
    thrust::device_vector<float> pointy(n);
    

    for (int i=0 ;i < n; i++){
        float x;
        float y;

        cin >> x;
        cin >> y;

        

        host_pointx[i] = x;
        host_pointy[i] = y;
        
    }

    pointx = host_pointx;
    pointy = host_pointy;



    thrust::device_vector<double> dist_matrix(n*n);

    dim3 blocks(ceil(n/32.0), ceil(n/32.0), 1);
    dim3 th(32, 32, 1);

    

    pre_calc<<<blocks,th>>>(
        thrust::raw_pointer_cast(pointx.data()),
        thrust::raw_pointer_cast(pointy.data()),
        thrust::raw_pointer_cast(dist_matrix.data()), 
        n
        );
    
    int all_seq_size = n*total_iter;
    
    thrust::host_vector<int> all_seq_host(all_seq_size);
    thrust::device_vector<int> all_seq(all_seq_size);
    thrust::device_vector<double> dis_calc(total_iter);





    for(int i=0;i<all_seq_size;i+=n){
        for (int j=0;j<n;j++){
            all_seq_host[i+j] = j;            
        }
    }
    
    //cout << "AAAAAAAAA" << endl;
    all_seq = all_seq_host;

    
    solver<<<max_blocks,max_th>>>(
        thrust::raw_pointer_cast(dist_matrix.data()),
        thrust::raw_pointer_cast(all_seq.data()),
        thrust::raw_pointer_cast(dis_calc.data()),
        n,
        total_iter
        );

    

    thrust::device_vector<double>::iterator iter = thrust::min_element(dis_calc.begin(), dis_calc.end());
      
    unsigned int position = iter - dis_calc.begin();
    double best = *iter;
      
    thrust::host_vector<double> host_best_seq(all_seq.begin()+position*n,all_seq.begin()+position*n+n);

    cout << best <<endl;
    for (auto i = host_best_seq.begin(); i != host_best_seq.end(); i++) {
        cout << *i << " ";
    }
    
    
    //DEBUG
    /*
    cout << best <<endl;
    for (auto i = all_seq.begin(); i != all_seq.end(); i++) {
        cout << *i << " "; // este acesso é lento! -- GPU
    }
    cout << endl;
    for (auto i = dis_calc.begin(); i != dis_calc.end(); i++) {
        cout << *i << " "; // este acesso é lento! -- GPU
    }
    cout << endl;
    for (auto i = host_best_seq.begin(); i != host_best_seq.end(); i++) {
        cout << *i << " "; // este acesso é lento! -- GPU
    }
    /**/
    

    return 0;
}